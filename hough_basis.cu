#include "hip/hip_runtime.h"
#include "common.hpp"
#include "hough_basis.hpp"
#include "math_functions.hpp"

template <typename Dtype>
__global__ void InitHoughBasis(const int H_, const int W_,
                               const int THETA_, const int RHO_,
                               const Dtype* sin_, const Dtype* cos_,
                               const int rho_min_, const int rho_step_,
                               Dtype* val_, int* ro_, int* ci_) {
  CUDA_KERNEL_LOOP(idx, H_*W_*THETA_) {
    const int hw = idx / THETA_;
    const int theta_i = idx % THETA_;
    const int h = hw / W_;
    const int w = hw % W_;
    const int ro = hw * THETA_;

    Dtype rho = h*sin_[theta_i] + w*cos_[theta_i];
    int rho_i = int( (rho-rho_min_)/rho_step_ );
    int ci = theta_i * RHO_ + rho_i;  // col idx
    val_[ro+theta_i] = Dtype(1);
    ci_[ro+theta_i] = ci;

    if (theta_i == 0) {
      ro_[hw] = ro;
      if (idx == H_*W_*THETA_-1) {
        ro_[hw+1] = ro + THETA_;
      }
    }
  }
}

template <typename Dtype>
void HoughBasis<Dtype>::Init_gpu() {
  const Dtype pi = std::acos(-1);
  SyncedMemory theta_(sizeof(Dtype) * THETA_);
  for (int theta_i = 0; theta_i < THETA_; theta_i++) {
    Dtype theta = theta_min_ + theta_i * theta_step_;
    ((Dtype*) theta_.mutable_cpu_data())[theta_i] = theta * pi / 180;
  }
  SyncedMemory sin_(sizeof(Dtype) * THETA_);
  SyncedMemory cos_(sizeof(Dtype) * THETA_);
  caffe_gpu_sincos(THETA_, (const Dtype*) theta_.gpu_data(),
                   (Dtype*) sin_.mutable_gpu_data(),
                   (Dtype*) cos_.mutable_gpu_data());
  InitHoughBasis<<<CAFFE_GET_BLOCKS(H_*W_*THETA_), CAFFE_CUDA_NUM_THREADS>>>(
      H_, W_, THETA_, RHO_, (const Dtype*) sin_.gpu_data(), (const Dtype*) cos_.gpu_data(),
      rho_min_, rho_step_, val_mutable_gpu_data(), ro_mutable_gpu_data(), ci_mutable_gpu_data());
}

template void HoughBasis<float>::Init_gpu();
template void HoughBasis<double>::Init_gpu();
